#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <fstream>
#include <vector>
#include <iostream>
#include <cassert>
#include <string>
#include <sstream>

#define MYASSERT(condition) if (!(condition)) { return; }


#define BLOCK_SIZE 32
#define HD __host__ __device__
#define WIDTH 512

HD float max_(float a, float b){ return (a < b) ? b : a; }
HD float min_(float a, float b){ return (a > b) ? b : a ;} 
static const float eps = 1e-8;
HD int clamp(int what, int low, int high);

#include "geometry.cuh"
#include "grid.cuh"

HD Vec3f trace(Vec3f rayorig, Vec3f raydir, Triangle* triangle_list, int tl_size);
HD Vec3f fast_trace(Ray& ray, GridAccel* newGridAccel, int isDebugThread);

__global__
void trace_kernel (float* params, Vec3f* image,GridAccel* d_newGridAccel, Triangle* triangle_list, Vec3f _u, Vec3f _v, Vec3f _w, Vec3f camerapos){

    //Pixel coordinates
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;	
		
    //Unpack parameters
    float invWidth = params[0], invHeight = params[1];
    float fov = params[2], _aspectratio = params[3];
    float angle = params[4];
    int tl_size = (int) params[5];

	//Unpack new parameters
	float width = params[6], height = params[7];
	float focal_distance = params[8];
	float aspectratio = params[9];
	Vec3f u = _u;
	Vec3f v = _v;
	Vec3f w = _w;

    Vec3f dir(0);
    dir = dir.add(w.negate().scale(focal_distance));
    float xw = aspectratio*(x - width/2.0 + 0.5)/width;
    float yw = (y - height/2.0 + 0.5)/height;
    dir = dir.add(u.scale(xw));
    dir = dir.add(v.scale(yw));
    dir.normalize();

    Ray ray(camerapos, dir, 0);

    //http://www.scratchapixel.com/lessons/3d-basic-rendering/introduction-to-ray-tracing/ray-tracing-practical-example

    //Trace ray
    //image[y*WIDTH + x] = trace(camerapos, dir, triangle_list, tl_size);
	image[y*WIDTH + x] = fast_trace(ray, d_newGridAccel, x == 275  && y == 240);	

}

Vec3f trace(Vec3f rayorig, Vec3f raydir, Triangle* triangle_list, int tl_size)
{

    //Ray triangle intersection
    float tnear = INFINITY,
          beta = INFINITY,
          gamma = INFINITY;

    const Triangle* triangle_near = NULL;
    for (unsigned int i = 0; i < tl_size; ++i) {
        float t0 = INFINITY,
              beta_ = INFINITY,
              gamma_ = INFINITY;
        if (triangle_list[i].rayTriangleIntersect(rayorig, raydir, t0, beta_, gamma_)) {

            if (t0 < tnear) {
                tnear = t0;
                triangle_near = &triangle_list[i];
                beta = beta_;
                gamma = gamma_;
            }
        }
    }
    if (!triangle_near){
        return Vec3f(0);
	}
 
    // Simple blinn phong shading
    Vec3f color(200.0);
    float kd = 0.3f;
    float ks = 0.5f;
    float spec_alpha = 4;

    // assume only 1 light over here.
    Vec3f light_pos(7, 7, -2);

    Vec3f poi = rayorig.add( raydir.scale(tnear) );
    Vec3f eye = rayorig.subtract(poi).normalize();  //raydir.negate();
    Vec3f l = poi.subtract(light_pos).normalize();
    Vec3f half = eye.add(l).normalize();
    Vec3f n = triangle_near->getNormal(poi).normalize();

    Vec3f diffuse = color.scale(kd * max_(float(0), n.dotProduct(l.normalize())));
    Vec3f specular = color.scale(ks * pow(max_(float(0), reflect(l,n).dotProduct(raydir.negate())), spec_alpha));
    Vec3f ambient = Vec3f(40.0);

    //return specular;
    // actual
    return diffuse.add(specular).add(ambient);

}

Vec3f fast_trace(Ray& ray, GridAccel* newGridAccel, int isDebugThread){

	Intersection* isect;
	Vec3f rayorig = ray.orig, raydir = ray.raydir;
	
	//Practically infinity
	Triangle triangle_near(Vec3f(100), Vec3f(100), Vec3f(100),Vec3f(100),Vec3f(100),Vec3f(100));
	float t0 = INFINITY;

	bool hitSomething = newGridAccel->Intersect(ray, isect, triangle_near, t0, isDebugThread);

    if (!hitSomething)
	    return Vec3f(0);
	
    // Simple blinn phong shading
    Vec3f color(200.0);
    float kd = 0.3f;
    float ks = 0.5f;
    float spec_alpha = 4;

    // assume only 1 light over here.
    Vec3f light_pos(7, 7, -2);

    Vec3f poi = rayorig.add( raydir.scale(t0) );
    Vec3f eye = rayorig.subtract(poi).normalize();  //raydir.negate();
    Vec3f l = light_pos.subtract(poi).normalize();
    Vec3f half = eye.add(l).normalize();
    Vec3f n = triangle_near.getNormal(poi).normalize();

   
    Vec3f diffuse = color.scale(kd * max_(float(0), n.dotProduct(l.normalize())));
    Vec3f specular = color.scale(ks * pow(max_(float(0), reflect(l,n).dotProduct(raydir.negate())), spec_alpha));
    Vec3f ambient = Vec3f(40.0f);

    // actual
    return diffuse.add(specular).add(ambient);

}

void render(std::vector<Triangle*> &triangle_list){

    //Define image size, calculate camera view parameters
    int width = WIDTH, height = WIDTH;
    Vec3f *image = new Vec3f[width * height], *pixel = image;
    float invWidth = 1 / float(width), invHeight = 1 / float(height);
    float _fov = 30, _aspectratio = width / float(height);
    float angle = tan(M_PI * 0.5 * _fov / 18);

    Vec3f camera_pos(0, 0, -3);
    Vec3f camera_target(0, 0, 0);
    Vec3f camera_up(0, -1, 0);
    float fov = 60;

    camera_up.normalize();
    Vec3f line_of_sight = camera_target.subtract(camera_pos);
    Vec3f w = line_of_sight.negate().normalize();
    Vec3f u = camera_up.crossProduct(w).normalize();
    Vec3f v = w.crossProduct(u).normalize();
    float focal_height = 1.0f;
    float aspectratio = float(width)/float(height);
    float focal_width = focal_height * aspectratio;
    float focal_distance = focal_height/(2.0 * tan(fov * M_PI/(180.0 * 2.0)));


    int tl_size = triangle_list.size();

    //Cuda events for timing data
    hipEvent_t c_start, c_stop, t_start, t_stop, k_start, k_stop;
    hipEventCreate(&c_start);
    hipEventCreate(&c_stop);
    hipEventCreate(&t_start);
    hipEventCreate(&t_stop);
    hipEventCreate(&k_start);
    hipEventCreate(&k_stop);
   
	//CHANGE THIS.
    float h_params[10] = {invWidth, invHeight, _fov, _aspectratio, angle, tl_size, width, height, focal_distance, aspectratio};

    Triangle* h_triangle_list = (Triangle*)malloc(tl_size*sizeof(Triangle));
    for(int i = 0;i<tl_size;i++)
        h_triangle_list[i] = *triangle_list[i];

    GridAccel* newGridAccel = new GridAccel(h_triangle_list, tl_size);	

	int totalVoxels = 1;
	for(int i=0;i<3;i++)
		totalVoxels *= newGridAccel->nVoxels[i];

    Voxel** h_voxels = (Voxel**)malloc(sizeof(Voxel*)*totalVoxels);

    //Parallel program begins	
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(width/dimBlock.x, height/dimBlock.y);

    float* d_params;
    Triangle* d_triangle_list;  
    Vec3f *d_image;
	GridAccel* d_newGridAccel;
	Voxel** d_voxels;

    //Copy parameters needed to set up camera view, triangle list
    hipMalloc(&d_params, 10*sizeof(float));    
    hipMalloc(&d_triangle_list, tl_size*sizeof(Triangle));
    hipMalloc(&d_image, width*height*sizeof(Vec3f));
	hipMalloc(&d_voxels, totalVoxels*sizeof(Voxel*));
	
	// Copying voxels to device memory
	int cnt = 0;

	for(int i = 0;i<totalVoxels;i++){

        if(newGridAccel->voxels[i] != NULL){
	
			Triangle* d_voxel_triangle_list;	
           	hipMalloc(&d_voxel_triangle_list, newGridAccel->voxels[i]->voxelListSize*sizeof(Triangle));
			hipMemcpy(d_voxel_triangle_list, newGridAccel->voxels[i]->triangleList, newGridAccel->voxels[i]->voxelListSize*sizeof(Triangle), hipMemcpyHostToDevice); 
	
			h_voxels[i] = (Voxel*)malloc(sizeof(Voxel));
			h_voxels[i]->voxelListSize = newGridAccel->voxels[i]->voxelListSize;	
			h_voxels[i]->triangleList = d_voxel_triangle_list;

			Voxel* d_voxel_elem;	
			hipMalloc(&d_voxel_elem, sizeof(Voxel));
			hipMemcpy(d_voxel_elem, h_voxels[i], sizeof(Voxel), hipMemcpyHostToDevice);
					          
			h_voxels[i] = d_voxel_elem;
	
        }
		cnt++;
    }

    hipMemcpy(d_voxels, h_voxels, totalVoxels*sizeof(Voxel*), hipMemcpyHostToDevice);

    //Allocate memory for GridAccel
    hipMalloc((void **)&d_newGridAccel, sizeof(GridAccel));
    newGridAccel->triangleList = d_triangle_list;
    newGridAccel->voxels = d_voxels;

    hipEventRecord(t_start);
    hipMemcpy(d_params, h_params, 10*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_triangle_list, h_triangle_list, tl_size*sizeof(Triangle), hipMemcpyHostToDevice);
	
	//Copy GridAccel
	hipMemcpy(d_newGridAccel, newGridAccel, sizeof(GridAccel), hipMemcpyHostToDevice);
	
	hipEventRecord(k_start);
    trace_kernel <<< dimGrid, dimBlock >>> (d_params, d_image, d_newGridAccel, d_triangle_list, u, v , w, camera_pos);
    hipEventRecord(k_stop);

    hipMemcpy(image, d_image, width*height*sizeof(Vec3f), hipMemcpyDeviceToHost);    
    hipEventRecord(t_stop);

    //Print timing data
    hipEventSynchronize(k_stop);
    float k_time = 0;
    hipEventElapsedTime(&k_time, k_start, k_stop);

    hipEventSynchronize(t_stop);
    float t_time = 0;
    hipEventElapsedTime(&t_time, t_start, t_stop);

    printf("GPU kernel time(ms): %f\n", k_time);
    printf("GPU total time(ms): %f\n", t_time);

    hipFree(d_image);
    hipFree(d_triangle_list);
    hipFree(d_params);
	hipFree(d_voxels);
	hipFree(d_newGridAccel);

    //Parallel program ends*/

    /*// Serial program begins
    hipEventRecord(c_start);
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x, ++pixel) {
            float xx = (2 * ((x + 0.5) * invWidth) - 1) * angle * aspectratio;
            float yy = (1 - 2 * ((y + 0.5) * invHeight)) * angle;
            Vec3f raydir(xx, yy, -2);
            raydir.normalize();
            *pixel = trace(Vec3f(0,0,-7), raydir, h_triangle_list, tl_size);
        }
        std::cout << y << "\n";
    }
    hipEventRecord(c_stop);

    //Print timing data
    hipEventSynchronize(c_stop);
    float c_time = 0;
    hipEventElapsedTime(&c_time, c_start, c_stop);

    printf("CPU total time(ms): %f\n", c_time);

    //Serial program ends*/

    //Write output to ppm file
    std::ofstream ofs("./blub0.ppm", std::ios::out | std::ios::binary);
    ofs << "P6\n" << width << " " << height << "\n255\n";
    for (int i = 0; i < width * height; ++i) {
        ofs << (unsigned char)(std::min(float(1), image[i].x/255)*255 ) <<
               (unsigned char)(std::min(float(1), image[i].y/255)*255 ) <<
               (unsigned char)(std::min(float(1), image[i].z/255)*255 );
    }
    ofs.close();

    //Free memory
    delete [] image;
    free(h_triangle_list);
	free(h_voxels);

}


int main(){

    std::ifstream objinfile("blub_triangulated.obj");

    std::string line;
    std::vector<Vec3f*> vertices;
    std::vector<Vec3f*> vertex_textures;
    std::vector<Triangle*> triangle_list;
    Triangle* triangle;

    //Reading obj file
    while(getline(objinfile, line)){

        std::istringstream iss(line);
        std::string type_;
        iss >> type_;
        std::string fv1, fv2, fv3;

	//Create list of vertices
        if (type_.compare("v") == 0){

            double a, b, c;
            iss >> a >> b >> c;
            vertices.push_back(new Vec3f(a, b, c));
        }
	//Create list of vertex textures
        else if (type_.compare("vt") == 0){
            double a, b;
            iss >> a >> b;
            vertex_textures.push_back(new Vec3f(a, b, 0));
        }
	//Create list of triangles
        else if (type_.compare("f") == 0){

            iss >> fv1 >> fv2 >> fv3;
            std::stringstream ssfv1(fv1);
            std::stringstream ssfv2(fv2);
            std::stringstream ssfv3(fv3);

            int v1, v2, v3;
            int vt1, vt2, vt3;
            ssfv1 >> v1;
            ssfv1.ignore();
            ssfv1 >> vt1;

            ssfv2 >> v2;
            ssfv2.ignore();
            ssfv2 >> vt2;

            ssfv3 >> v3;
            ssfv3.ignore();
            ssfv3 >> vt3;

            Vec3f *vertex1 = vertices[v1 - 1];
            Vec3f *vertex2 = vertices[v2 - 1];
            Vec3f *vertex3 = vertices[v3 - 1];

            triangle = new Triangle(*vertices[v1-1], *vertices[v2-1], *vertices[v3-1],
                                    *vertex_textures[vt1-1], *vertex_textures[vt2-1], *vertex_textures[vt3-1]);

            triangle_list.push_back(triangle);
        }
    }

    render(triangle_list);

    return 0;
}


HD double det(double a1, double a2, double a3,
            double b1, double b2, double b3,
            double c1, double c2, double c3)
{
    double t1 = a1 * (b2*c3 - b3*c2);
    double t2 = a2 * (b1*c3 - b3*c1);
    double t3 = a3 * (b1*c2 - b2*c1);
    return t1 - t2 + t3;
}

HD int clamp(int what, int low, int high)
{
    if (what < low) return low;
    if (what > high) return high;
    return what;
}

