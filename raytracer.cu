
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <fstream>
#include <vector>
#include <iostream>
#include <cassert>
#include <string>
#include <sstream>
//#include <png++/png.hpp>

static const float eps = 1e-8;
double det(double a1, double a2, double a3,
            double b1, double b2, double b3,
            double c1, double c2, double c3);
int clamp(int what, int low, int high);

template<typename T>
class Vec3{
public:

    T x, y, z;
    //Constructors
    Vec3() : x(T(0)), y(T(0)), z(T(0)) {}
    Vec3(T val) : x(val), y(val), z(val) {}
    Vec3(T xval, T yval, T zval) : x(xval), y(yval), z(zval) {}

    Vec3& normalize(){
        T nor2 = length2();
        if (nor2 > 0) {
            T nor_inv = 1 / sqrt(nor2);
            x *= nor_inv, y *= nor_inv, z *= nor_inv;
        }
        return *this;
    }

    T dotProduct(const Vec3<T> &v) const {
        return x * v.x + y * v.y + z * v.z;
    }

    Vec3<T> crossProduct(const Vec3<T> &v) const {

        T tmpX = y * v.z - z * v.y;
        T tmpY = z * v.x - x * v.z;
        T tmpZ = x * v.y - y * v.x;
        return Vec3<T>(tmpX, tmpY, tmpZ );
    }

    T length2(){
        return x * x + y * y + z * z;
    }
    T length(){
        return sqrt(length2());
    }

    Vec3<T> scale(const T &f) const {
        return Vec3<T>(x * f, y * f, z * f);
    }

    Vec3<T> multiply(const Vec3<T> &v) const {
        return Vec3<T>(x * v.x, y * v.y, z * v.z);
    }

    Vec3<T> subtract(const Vec3<T> &v) const {
        return Vec3<T>(x - v.x, y - v.y, z - v.z);
    }

    Vec3<T> add(const Vec3<T> &v) const {
        return Vec3<T>(x + v.x, y + v.y, z + v.z);
    }

    Vec3<T> negate() const {
        return Vec3<T>(-x, -y, -z);
    }

    //Helper function to format display
    friend std::ostream & operator << (std::ostream &os, const Vec3<T> &v){
        os << "(" << v.x << " " << v.y << " " << v.z << ")";
        return os;
    }
};

typedef Vec3<float> Vec3f;

Vec3f reflect(const Vec3f &I, const Vec3f &N){
	return I.subtract(N.scale(2*I.dotProduct(N)));
}

class Triangle{
public:

    Vec3f v0, v1, v2;
    Vec3f tv0, tv1, tv2; // texture coordinates of vertices

     Triangle(
        const Vec3f &v_0,
        const Vec3f &v_1,
        const Vec3f &v_2,
        const Vec3f &tv_0,
        const Vec3f &tv_1,
        const Vec3f &tv_2) :
        v0(v_0), v1(v_1), v2(v_2),
        tv0(tv_0), tv1(tv_1), tv2(tv_2)
     { /* empty */ }

    //Not our stuff yet.
    bool rayTriangleIntersect(const Vec3f &orig, const Vec3f &dir, float &t, float &beta, float &gamma){

        double A = det(
                    v0.x - v1.x, v0.x - v2.x, dir.x,
                    v0.y - v1.y, v0.y - v2.y, dir.y,
                    v0.z - v1.z, v0.z - v2.z, dir.z
                    );

        double t_ = det(
                    v0.x - v1.x, v0.x - v2.x, v0.x - orig.x,
                    v0.y - v1.y, v0.y - v2.y, v0.y - orig.y,
                    v0.z - v1.z, v0.z - v2.z, v0.z - orig.z
                    );
        t_ = t_ / A;

        double beta_ = det(
                    v0.x - orig.x, v0.x - v2.x, dir.x,
                    v0.y - orig.y, v0.y - v2.y, dir.y,
                    v0.z - orig.z, v0.z - v2.z, dir.z
                    );
        beta_ = beta_ / A;

        double gamma_ = det(
                    v0.x - v1.x, v0.x - orig.x, dir.x,
                    v0.y - v1.y, v0.y - orig.y, dir.y,
                    v0.z - v1.z, v0.z - orig.z, dir.z
                    );
        gamma_ = gamma_ / A;

        if (beta_ > 0 && gamma_ > 0 && beta_ + gamma_ < 1)
        {
            t = t_;
            beta = beta_;
            gamma = gamma_;
            return true;
        }
        return false;

    }

    Vec3f getNormal(Vec3f point) const
    {
        // from http://math.stackexchange.com/a/137551
        Vec3f p = point.subtract(v1);
        Vec3f q = v0.subtract(v2);

        // use point here. !!

        return Vec3f(
                    p.y * q.z - p.z * q.y,
                    -1 * (p.x*q.z - p.z * q.x),
                    p.x*q.y - p.y*q.x
                    );
    }
};

void print_vec3f(const char *label, const Vec3f &v)
{
//    std::cout << label << " [" << v.x << "," << v.y << "," << v.z << "]";
}

Vec3f trace(Vec3f rayorig, Vec3f raydir,
            const std::vector<Triangle*> &triangle_list)
{
    float tnear = INFINITY,
          beta = INFINITY,
          gamma = INFINITY;

    const Triangle* triangle_near = NULL;
    for (unsigned int i = 0; i < triangle_list.size(); ++i) {
        float t0 = INFINITY,
              beta_ = INFINITY,
              gamma_ = INFINITY;
        if (triangle_list[i]->rayTriangleIntersect(rayorig, raydir, t0, beta_, gamma_)) {
            if (t0 < tnear) {
                tnear = t0;
                triangle_near = triangle_list[i];
                beta = beta_;
                gamma = gamma_;
            }
        }
    }
    if (!triangle_near)
        return Vec3f(0);
 
    // Simple blinn phong shading
    Vec3f color(fabs(200));
    float kd = 5.0f;
    float ks = 0.5f;
    float spec_alpha = 4;

    // assume only 1 light over here.
    Vec3f light_pos(5, 5, -10);

    Vec3f poi = rayorig.add( raydir.scale(tnear) );
    Vec3f eye = rayorig.subtract(poi).normalize();  //raydir.negate();
    Vec3f l = poi.subtract(light_pos).normalize();
    Vec3f half = eye.add(l).normalize();
    Vec3f n = triangle_near->getNormal(poi).normalize();

    print_vec3f("eye", eye);
    print_vec3f(" poi", poi);
    print_vec3f(" l", l);
    print_vec3f(" half", half);
    print_vec3f(" n", n);

    Vec3f diffuse = color.scale(kd * std::max(float(0), n.dotProduct(l.normalize())));
    Vec3f specular = color.scale(ks * pow(std::max(float(0), reflect(l,n).dotProduct(raydir.negate())), spec_alpha));
    Vec3f ambient = Vec3f(50);

    print_vec3f(" diffuse", diffuse);
    print_vec3f(" specular", specular);

//    std::cout << std::endl;

    //return l.normalize().scale(100.0f);

    // debugging
    //  return eye.scale(10.0f);

    return specular;
    // actual
    return diffuse.add(specular).add(ambient);

}

void render(const std::vector<Triangle*> &triangle_list){

    int width = 1024, height = 1024;
    Vec3f *image = new Vec3f[width * height], *pixel = image;
    float invWidth = 1 / float(width), invHeight = 1 / float(height);
    float fov = 30, aspectratio = width / float(height);
    float angle = tan(M_PI * 0.5 * fov / 180.);

    // Trace rays
    int cnt = 0;
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x, ++pixel) {
            float xx = (2 * ((x + 0.5) * invWidth) - 1) * angle * aspectratio;
            float yy = (1 - 2 * ((y + 0.5) * invHeight)) * angle;
            Vec3f raydir(xx, yy, -2);
            raydir.normalize();
            *pixel = trace(Vec3f(0,0,-7), raydir, triangle_list);
        }
        std::cout << y << "\n"; cnt++;
    }
    // Save result to a PPM image (keep these flags if you compile under Windows)
    std::ofstream ofs("./trial11.ppm", std::ios::out | std::ios::binary);
    ofs << "P6\n" << width << " " << height << "\n255\n";
    for (int i = 0; i < width * height; ++i) {
        ofs << (unsigned char)(std::min(float(1), image[i].x/255)*255 ) <<
               (unsigned char)(std::min(float(1), image[i].y/255)*255 ) <<
               (unsigned char)(std::min(float(1), image[i].z/255)*255 );
    }
    ofs.close();
    delete [] image;
}


int main(){

    std::ifstream objinfile("input.obj");

    std::string line;
    std::vector<Vec3f*> vertices;
    std::vector<Vec3f*> vertex_textures;
    std::vector<Triangle*> triangle_list;
    Triangle* triangle;

    while(getline(objinfile, line)){

        std::istringstream iss(line);
        std::string type_;
        iss >> type_;
        std::string fv1, fv2, fv3;
        if (type_.compare("v") == 0){

            double a, b, c;
            iss >> a >> b >> c;
            vertices.push_back(new Vec3f(a, b, c));
        }
        else if (type_.compare("vt") == 0){
            double a, b;
            iss >> a >> b;
            vertex_textures.push_back(new Vec3f(a, b, 0));
        }
        else if (type_.compare("f") == 0){

            iss >> fv1 >> fv2 >> fv3;
            std::stringstream ssfv1(fv1);
            std::stringstream ssfv2(fv2);
            std::stringstream ssfv3(fv3);

            int v1, v2, v3;
            int vt1, vt2, vt3;
            ssfv1 >> v1;
            ssfv1.ignore();
            ssfv1 >> vt1;

            ssfv2 >> v2;
            ssfv2.ignore();
            ssfv2 >> vt2;

            ssfv3 >> v3;
            ssfv3.ignore();
            ssfv3 >> vt3;

            Vec3f *vertex1 = vertices[v1 - 1];
            Vec3f *vertex2 = vertices[v2 - 1];
            Vec3f *vertex3 = vertices[v3 - 1];

            triangle = new Triangle(*vertices[v1-1], *vertices[v2-1], *vertices[v3-1],
                                    *vertex_textures[vt1-1], *vertex_textures[vt2-1], *vertex_textures[vt3-1]);

            triangle_list.push_back(triangle);
        }
    }

    render(triangle_list);

    return 0;
}


double det(double a1, double a2, double a3,
            double b1, double b2, double b3,
            double c1, double c2, double c3)
{
    double t1 = a1 * (b2*c3 - b3*c2);
    double t2 = a2 * (b1*c3 - b3*c1);
    double t3 = a3 * (b1*c2 - b2*c1);
    return t1 - t2 + t3;
}

int clamp(int what, int low, int high)
{
    if (what < low) return low;
    if (what > high) return high;
    return what;
}

